#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
// Kernel function to add the elements of two arrays
__global__
void add(int n, float* x, float* y)
{
    for (int i = 0; i < n; i++)
        y[i] = x[i] + y[i];
}

int main(void)
{
    int N = 1 << 20;
    float* x, * y;

    // Allocate Unified Memory � accessible from CPU or GPU
    hipMallocManaged(&x, N * sizeof(float));
    hipMallocManaged(&y, N * sizeof(float));

    // initialize x and y arrays on the host
    for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    // Run kernel on 1M elements on the GPU
    add << <1, 1 >> > (N, x, y);

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    /*
    Just one more thing: I need the CPU to wait until the kernel is done before it accesses the results 
    (because CUDA kernel launches don�t block the calling CPU thread). 
    To do this I just call hipDeviceSynchronize() before doing the final error checking on the CPU.
    */

    // Check for errors (all values should be 3.0f)
    float maxError = 0.0f;
    for (int i = 0; i < N; i++)
        maxError = fmax(maxError, fabs(y[i] - 3.0f));
    std::cout << "Max error: " << maxError << std::endl;

    // Free memory
    hipFree(x);
    hipFree(y);

    return 0;
}
/*

This is only a first step, because as written, this kernel is only correct for a single thread, 
since every thread that runs it will perform the add on the whole array. 
Moreover, there is a race condition since multiple parallel threads would both read and write the same locations.

*/