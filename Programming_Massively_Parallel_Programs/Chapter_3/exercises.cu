#include "hip/hip_runtime.h"
//In the chapter we produced a matMul kernel which has each thread produce one output matrix element:
__global__
void MatrixMulKernel(float* M, float* N, float* P, int Width) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int	col = blockIdx.x * blockDim.x + threadIdx.x;

	if ((row < Width) && (col < width)) {
		float Pvalue = 0;
		for (int k = 0; k < Width; ++k) {
			//Beginning Element of row 1 is M[1*Width] accessing the kth element of mth rowth row is M[row*Width+k]
			//Beginning element of colth column is the colth element of row 0 which is N[col]
				//Kth element of the colth column is N[k*Width+col] which means we skip over whole rows
			Pvalue += M[row * Width + k] * N[k * Width + col];
		}
		P[row * Width + col] = Pvalue;
	}
}

//Exericse 1.A: Write a kernel that has each thread produce one output matrix row:
__global__
void MatrixMulKernelRow(float* M, float* N, float* P, int Width) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	
	if (row < Width) {
		for (int col = 0; col < Width; ++col) {
			//Would compute P[row][col]
			float Pvalue = 0;
			for (int k = 0; k < Width; ++k) {

				Pvalue += M[row * Width + k] * N[k * Width + col];
			}
			P[row * Width + col] = Pvalue;
		}	
	}
}
//Launch Configs:
// Each thread handles a single row
int blockSize = 4;  // Number of threads per block, originally 4 th/bl with 4 blocks now still 4 threads per block but just a single block
int numBlocks = (Width + blockSize - 1) / blockSize;  // Total blocks needed, technically 1

dim3 blockDim(blockSize);  // Threads in y-dimension
dim3 gridDim(numBlocks);   // Blocks in y-dimension

MatrixMulKernelRowWise << <gridDim, blockDim >> > (M, N, P, Width);

//Exericse 1.B: Write a kernel that has each thread produce one output matrix col:
__global__
void MatrixMulKernelCol(float* M, float* N, float* P, int Width) {
	int	col = blockIdx.x * blockDim.x + threadIdx.x;

	if (col < Width) {
		for (int row = 0; row < Width; ++row) {
			//Would compute P[row][col]
			float Pvalue = 0;
			for (int k = 0; k < Width; ++k) {

				Pvalue += M[row * Width + k] * N[k * Width + col];
			}
			P[row * Width + col] = Pvalue;
		}
	}
}
//Launch Configs:
// Each thread handles a single column
int blockSize = 4;  // Number of threads per block, originally 4 th/bl now still 4 threads per block
int numBlocks = (Width + blockSize - 1) / blockSize;  // Total blocks needed, technically 1

dim3 blockDim(blockSize);  // Threads in x-dimension
dim3 gridDim(numBlocks);   // Blocks in x-dimension

MatrixMulKernelColWise << <gridDim, blockDim >> > (M, N, P, Width);


//Exericse 1.C: What are the pro/cons of Col-wise and Row-wise?
/*
--Row_wise--
P:
	More memory efficient as threads in a warp/group would be accessing consecutive memory locations (speed) -- coalesced memory
		Row access for M but not for N so memory inefficient for that matrix (Stride access and thus non-coalesced)
	Row-wise is generally how most things are laid out (conceptually easier for me at least haha)
C:
	Non-coalesced for N, and divergence of work efforts depending on how elements are laid out in a matrix, some threads may do more work than others!
--Col_wise--
P:
	Memory eff for N (col-wise) but not for M (row-wise)
	Wide v tall matrices
C:
	Uneven work/thread divergence (like Row_wise).


*/

//Exercise 2:
/*

A matmul takes an input mat B and vector C and produces one output vector A,
Each element of the output vector A is the dot product of one row of the input mat B and C that is A[i]= EjB[i][j] + C[j]
We'll only handle square matrices 
Write a kernel with four params: point to the out mat, pointer to the in mat, point to the in vec and number of elements in each dim
	Use one thread to calculate an output vector element

*/
void MatVecMulKernel(float* OUT, float* IN_M, float* IN_V, int N) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	
	if (row < N){
		float Pvalue = 0;
		for (int k = 0; k < N; ++k) {
			Pvalue += IN_M[row * N + k] * IN_V[k];
		}
	OUT[row] = Pvalue;
	}
}

//Exercise 3:
/*

Consider this cuda kernel

__global__
void foo_kernel(float* a, float* b, unsigned int M, unsigned int N){
	unsigned int row = blockIdx.y*blockDim.y+threadIdx.y;
	unsigned int col = blockIdx.x*blockDim.x+threadIdx.x;
	if( row < M && col < N){
		b[row*N + col] = a[row*N+col]/2.1f + 4.8f;
	}
}

void foo(float* a_d, float* b_d){
	unsigned int M = 150;
	unsigned int N = 300;
	dim3 bd(16, 32);
	dim3 gd((N-1)/16 + 1, (M-1)/32 + 1);
	foo_kernel<<<gd, bd>>>(a_d, b_d, M, N);
}

\\\Questions///
1. Threads per Block?
	.X = 16 & .y = 32 so 16*32 = 512 threads per block
2. Number of threads in the grid?
	grid.x = (300-1)/16 + 1 == 19 -- 20
	grid.y = (150-1)/32 + 1 == 5 -- 6
	19 * 5 = 95 Blocks * 512 = 48640
	so 20*6 = 120 Blocks, 120*512 = 61,440 Threads in the grid
3. Number of blocks in the grid?
	120
4. Number of threads that execute code on line 5?
	Line 5: b[row*N + col] = a[row*N+col]/2.1f + 4.8f;
	..Well.. Line 5 is gatekept by: if( row < M && col < N) and M == 150, N=300 and 150*300 = 45K!

*/

//Exercise 4:
/*

W = 400, H = 500, (200K elements) specifiy the array index of the element at row 20 and col 10.
A. In row-major order: M[20*400+10] == 8010th element
B. In Col-major order: M[20+10*500] == 5020th element

*/

//Exercise 5:
/*

3D Tensor now, W=400, H=500, D=300, laying it out as a 1D array in row-major order whats the index of X=10 y=20, Z=5? (Row 20, Col 10, Depth 5
60M elements

P[plane*m*n+row*m+col]

P[5*400*500+20*400+10] = 1,008,010th element
*/